

#include <hip/hip_runtime.h>
#include "hip/hip_fp16.h"
#include <cuda_pipeline.h>
#include <stdio.h>
#include <cmath>
#include <tuple>

#define div_ru(a, b) (a + b - 1) / b

#define WARP_SIZE 32

#define CUDA_DEVICE_INLINE __device__ __forceinline__

CUDA_DEVICE_INLINE float warpReduceSumAllThreads(float val)
{
    for (int offset = warpSize / 2; offset > 0; offset /= 2)
    {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return __shfl_sync(0xffffffff, val, 0);
}

struct __align__(8) half4
{
    half vals[4];
};

// the vllm stuff is copied from the vllm repo
// (https://github.com/vllm-project/vllm/blob/3711811b1d2956e83e626c72f0e1607f2dfbc8fb/csrc/cuda_compat.h)
//  and (https://github.com/vllm-project/vllm/blob/3711811b1d2956e83e626c72f0e1607f2dfbc8fb/csrc/attention/attention_kernels.cu#L45)
#define VLLM_SHFL_XOR_SYNC(var, lane_mask) __shfl_xor_sync(uint32_t(-1), var, lane_mask)
#define VLLM_SHFL_SYNC(var, src_lane) __shfl_sync(uint32_t(-1), var, src_lane)

template <int NUM_WARPS>
CUDA_DEVICE_INLINE float block_sum(float *red_smem, float sum)
{
    // Decompose the thread index into warp / lane.
    int warp = threadIdx.x / WARP_SIZE;
    int lane = threadIdx.x % WARP_SIZE;

    // Compute the sum per warp.
#pragma unroll
    for (int mask = WARP_SIZE / 2; mask >= 1; mask /= 2)
    {
        sum += VLLM_SHFL_XOR_SYNC(sum, mask);
    }

    // Warp leaders store the data to shared memory.
    if (lane == 0)
    {
        red_smem[warp] = sum;
    }

    // Make sure the data is in shared memory.
    __syncthreads();

    // The warps compute the final sums.
    if (lane < NUM_WARPS)
    {
        sum = red_smem[lane];
    }

    // Parallel reduction inside the warp.
#pragma unroll
    for (int mask = NUM_WARPS / 2; mask >= 1; mask /= 2)
    {
        sum += VLLM_SHFL_XOR_SYNC(sum, mask);
    }

    // Broadcast to other threads.
    return VLLM_SHFL_SYNC(sum, 0);
}

template <int headSize, int numQheads, int numKVheads, int numBlocks>
__global__ void cache_attn(const half *q,  // [batchSize, numQheads, 1, headSize]
                           const half *k,  // [batchSize, numKVheads, 1, headSize]
                           const half *v,  // [batchSize, numKVheads, 1, headSize]
                           half *cache,    // [numBlocks, 2, batchSize, numKVheads, totalSeq, headSize]
                           const int *pos, // [batchSize]
                           const int *pc,  // [batchSize]
                           half *out,      // [batchSize, numQheads, 1, headSize]
                           int batchSize,
                           int totalSeq,
                           int layer_index)
{

    const int batchIndex = blockIdx.x;
    const int headIndex = blockIdx.y;

    const int kvHeadIndex = headIndex / (numQheads / numKVheads);

    const int qOffset = batchIndex * numQheads * headSize + headIndex * headSize;
    const int kvOffset = batchIndex * numKVheads * headSize + kvHeadIndex * headSize;
    const int outOffset = qOffset;

    const int thisSeqLen = pos[batchIndex];

    const int cumulSeqLen = pc[batchIndex];

    // const int kCacheOffset = layer_index * 2 * numKVheads * totalSeq * headSize + 0 * numKVheads * totalSeq * headSize + kvHeadIndex * totalSeq * headSize;
    // const int vCacheOffset = layer_index * 2 * numKVheads * totalSeq * headSize + 1 * numKVheads * totalSeq * headSize + kvHeadIndex * totalSeq * headSize;

    const int kCacheOffset = /* block stride */ layer_index * 2 * batchSize * numKVheads * totalSeq * headSize +
                             /* kv stride    */ 0 * batchSize * numKVheads * totalSeq * headSize +
                             /* batch stride */ batchIndex * numKVheads * totalSeq * headSize +
                             /* head stride  */ kvHeadIndex * totalSeq * headSize;

    const int vCacheOffset = /* block stride */ layer_index * 2 * batchSize * numKVheads * totalSeq * headSize +
                             /* kv stride    */ 1 * batchSize * numKVheads * totalSeq * headSize +
                             /* batch stride */ batchIndex * numKVheads * totalSeq * headSize +
                             /* head stride  */ kvHeadIndex * totalSeq * headSize;

    __shared__ half sharedQ[headSize];

    constexpr int NUM_WARPS = headSize / WARP_SIZE;

    __shared__ float blockReduction[2 * NUM_WARPS];

    if (thisSeqLen == 0)
    {
        return;
    }

    sharedQ[threadIdx.x] = __float2half(__half2float(q[qOffset + threadIdx.x]) * 1.44269504 * (1.f / sqrt(headSize)));

    __syncthreads();

    float acc = 0.0f;

    float mi = -50000.f;
    float li = 0.f;

    for (int s = 0; s < thisSeqLen; s++)
    {

        half kItem = (cache[kCacheOffset + s * headSize + threadIdx.x]);

        float thisQk = __half2float(sharedQ[threadIdx.x]) * __half2float(kItem);

        float qk = block_sum<NUM_WARPS>(&blockReduction[NUM_WARPS], thisQk);

        // cache[kCacheOffset + s * headSize + threadIdx.x] = kItem;

        float mi_new = max(mi, qk);

        float alpha = exp2f(mi - mi_new);

        float p = exp2f(qk - mi_new);

        half vitem = cache[vCacheOffset + s * headSize + threadIdx.x];

        acc *= alpha;

        acc += p * __half2float(vitem);

        li = li * alpha + p;
        mi = mi_new;

        // cache[vCacheOffset + (s + batchIndex) * headSize + threadIdx.x] = vitem;
    }

    half finalKItem = k[kvOffset + threadIdx.x];

    float qk = __half2float(sharedQ[threadIdx.x]) * __half2float(finalKItem);

    qk = block_sum<NUM_WARPS>(&blockReduction[NUM_WARPS], qk);

    float mi_new = max(mi, qk);

    float alpha = exp2f(mi - mi_new);

    float p = exp2f(qk - mi_new);

    half finalVItem = v[kvOffset + threadIdx.x];

    acc *= alpha;

    acc += p * __half2float(finalVItem);

    li = li * alpha + p;

    cache[kCacheOffset + (thisSeqLen + 0) * headSize + threadIdx.x] = finalKItem;
    cache[vCacheOffset + (thisSeqLen + 0) * headSize + threadIdx.x] = finalVItem;

    out[outOffset + threadIdx.x] = __float2half(acc / li);
}

#define LAUNCH_KERNEL_IF_CONDITION(headDim, numHeads, numKVHeads, numBlocks)                                                                                                 \
    else if (num_heads == numHeads && num_kv_heads == numKVHeads && head_dim == headDim && num_blocks == numBlocks)                                                          \
    {                                                                                                                                                                        \
        auto kernelFunc = cache_attn<headDim, numHeads, numKVHeads, numBlocks>;                                                                                              \
                                                                                                                                                                             \
        constexpr unsigned int smem = 0;                                                                                                                                     \
                                                                                                                                                                             \
        hipFuncSetAttribute(                                                                                                                                                \
            reinterpret_cast<const void*>(kernelFunc),                                                                                                                                                      \
            hipFuncAttributeMaxDynamicSharedMemorySize,                                                                                                                     \
            smem);                                                                                                                                                           \
                                                                                                                                                                             \
        dim3 blocks_per_grid(batchSize, numHeads);                                                                                                                           \
        constexpr dim3 threads_per_block(headDim);                                                                                                                           \
        kernelFunc<<<blocks_per_grid, threads_per_block, smem, stream>>>(Q_ptr, K_ptr, V_ptr, Cache_ptr, P_ptr, PCumulative_ptr, Out_ptr, batchSize, totalSeq, layer_index); \
        return;                                                                                                                                                              \
    }

void wrapper(void *q, void *k, void *v, void *c, void *p, void *pc, void *o, const int batchSize, const int totalSeq, const int layer_index, const int num_heads, const int num_kv_heads, const int head_dim, const int num_blocks, hipStream_t stream)
{

    const half *Q_ptr = reinterpret_cast<const half *>(q);
    const half *K_ptr = reinterpret_cast<const half *>(k);
    const half *V_ptr = reinterpret_cast<const half *>(v);
    half *Cache_ptr = reinterpret_cast<half *>(c);
    const int *P_ptr = reinterpret_cast<const int *>(p);
    const int *PCumulative_ptr = reinterpret_cast<const int *>(pc);
    half *Out_ptr = reinterpret_cast<half *>(o);

    if (false)
    {
    }
    LAUNCH_KERNEL_IF_CONDITION(128, 32, 8, 32)
    LAUNCH_KERNEL_IF_CONDITION(80, 32, 32, 32)
    LAUNCH_KERNEL_IF_CONDITION(128, 64, 8, 80)
}